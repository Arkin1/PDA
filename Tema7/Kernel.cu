
#include "hip/hip_runtime.h"


#include<iostream>
#include<algorithm>
using namespace std;

#define N 5

int D[N][N] = {
	0,3,9,8,3,
	5,0,1,4,2,
	6,6,0,4,5,
	2,9,2,0,7,
	7,9,3,2,0,
};

int k;

__global__ void MatAdd(int D[N][N])
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	D[i][j] = min(D[i][j], D[i][k] + D[k][j]);
}


int main()
{
	int numBlocks = 1;

	dim3 threadsPerBlock(N, N);

	
   
	for (k = 1; k <= N; ++k)
	{
		MatAdd<<<numBlocks, threadsPerBlock>>>(D);
	}
}
